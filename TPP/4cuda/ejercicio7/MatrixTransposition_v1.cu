
/****************************************
 * CUDA kernel for transposing matrices *
 *  * Puede dar error en la version de gpu, revisar el Makefile la siguiente linea ARCH=-arch sm_20, puede ser sm_35      *
 ****************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }

#define	A(i,j)		A[ (j) + ((i)*(n)) ]
#define	B(i,j)		B[ (j) + ((i)*(m)) ]
#define	B_cpu(i,j) 	B_cpu[ (j) + ((i)*(m)) ]
#define	B_gpu(i,j) 	B_gpu[ (j) + ((i)*(m)) ]
#define	d_A(i,j) 	d_A[ (j) + ((i)*(n)) ]
#define	d_B(i,j) 	d_B[ (j) + ((i)*(m)) ]

__global__ void compute_kernel( unsigned int m, unsigned int n, float *d_A, float *d_B ) {
    /* Index of thread in x dimension */
    /* Index of thread in y dimension */
    int x = threadIdx.x;
    int y = threadIdx.y;
    /* Global index to a matrix row (i) */
    /* Global index to a matrix col (j) */
    int i = x + blockIdx.x + blockDim.x;
    int j = y + blockIdx.y + blockDim.y;
    /* Copy element A(i,j) into B(j,i) to form the transposed matrix */
    if (i < m && j < n)
    {
    d_B( j, i ) = d_A( i, j );
    }
}

int cu_transpose( unsigned int m, unsigned int n, unsigned int block_size, float *h_A, float *h_B  ) {

  // Allocate device memory
  unsigned int mem_size = m * n * sizeof(float);
  float *d_A, *d_B;
  CUDA_SAFE_CALL( hipMalloc((void **) &d_A, mem_size ) );
  CUDA_SAFE_CALL( hipMalloc((void **) &d_B, mem_size ) );

  // Copy host memory to device 
  CUDA_SAFE_CALL( hipMemcpy( d_A, h_A, mem_size, hipMemcpyHostToDevice ) );

  // Calculate blocks grid size
  int blocks_col = (int) ceil( (float) n / (float) block_size );
  int blocks_row = (int) ceil( (float) m / (float) block_size );

  // Execute the kernel
  dim3 dimGrid( blocks_col, blocks_row );
  dim3 dimBlock( block_size, block_size );
  compute_kernel<<< dimGrid, dimBlock >>>( m, n, d_A, d_B );

  // Copy device memory to host 
  CUDA_SAFE_CALL( hipMemcpy( h_B, d_B, mem_size, hipMemcpyDeviceToHost ) );

  // Deallocate device memory
  CUDA_SAFE_CALL( hipFree(d_A) );
  CUDA_SAFE_CALL( hipFree(d_B) );

  return EXIT_SUCCESS;
}
 
int transpose( unsigned int m, unsigned int n, float *A, float *B ) {

  unsigned int i, j;
  for( i=0; i<m; i++ ) {
    for( j=0; j<n; j++ ) {
      B( j, i ) = A( i, j );
    }
  }
  return EXIT_SUCCESS;

}

void printMatrix( unsigned int m, unsigned int n, float *A ) {
  int i, j;
  for( i=0; i<m; i++ ) {
    for( j=0; j<n; j++ ) {
      printf("%8.1f",A(i,j));
    }
    printf("\n");
  }
}

int main( int argc, char *argv[] ) {
  unsigned int m, n;
  unsigned int block_size;
  unsigned int i, j;

  /* Generating input data */
  if( argc<4 ) {
    printf("Usage: %s n_rows n_cols block_size \n",argv[0]);
    exit(-1);
  }
  sscanf(argv[1],"%d",&m);
  sscanf(argv[2],"%d",&n);
  sscanf(argv[3],"%d",&block_size);
  float *A = (float *) malloc( m*n*sizeof(float) );
  printf("%s: Generating a random matrix of size %dx%d...\n",argv[0],m,n);
  for( i=0; i<m; i++ ) {
    for( j=0; j<n; j++ ) {
      A( i, j ) = 2.0f * ( (float) rand() / RAND_MAX ) - 1.0f;
    }
  }
  float *B_cpu = (float *) malloc( m*n*sizeof(float) );
  float *B_gpu = (float *) malloc( m*n*sizeof(float) );

  printf("%s: Transposing matrix A into B in CPU...\n",argv[0]);
  transpose( m, n, A, B_cpu );

  printf("%s: Transposing matrix A into B in GPU...\n",argv[0]);
  cu_transpose( m, n, block_size, A, B_gpu );

  /* Check for correctness */
  float error = 0.0f;
  for( i=0; i<n; i++ ) {
    for( j=0; j<m; j++ ) {
      error += fabs( B_gpu( i, j ) - B_cpu( i, j ) );
    }
  }
  printf("Error CPU/GPU = %.3e\n",error);
  
  free(A);
  free(B_cpu);
  free(B_gpu);
  
}

