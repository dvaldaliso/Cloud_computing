
/************************************************
 * Simple CUDA example to transfer data CPU-GPU *
 ************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_SAFE_CALL( call ) {                                         \
 hipError_t err = call;                                                 \
 if( hipSuccess != err ) {                                              \
   fprintf(stderr,"CUDA: error occurred in cuda routine. Exiting...\n"); \
   exit(err);                                                            \
 } }

#define	A(i,j)		A[ (j) + ((i)*(n)) ]
#define	B(i,j) 		B[ (j) + ((i)*(n)) ]

int main( int argc, char *argv[] ) {
  unsigned int m, n;
  unsigned int i, j;

  /* Generating input data */
  if( argc<3 ) {
    printf("Usage: %s rows cols \n",argv[0]);
    exit(-1);
  }
  sscanf(argv[1],"%d",&m);
  sscanf(argv[2],"%d",&n);

  /* STEP 1: Allocate memory for two m-by-n matrices called A and B in the host */
  float *A, *B;
  A = (float*) malloc(m*n*sizeof(float));
  B = (float*) malloc(m*n*sizeof(float));

  /* STEP 2: Fill matrix A with real random values between -1.0 and 1.0 */
  for (i=0; i < m; i++) { 
      for (j=0; j < n; j++) { 
          A(i,j) = (2.0f * (float) rand() / RAND_MAX ) - 1.0f; 
      }
   }
  /* STEP 3: Allocate memory for three m-by-n matrices into the device memory */
  float *d_A, *d_B;
  CUDA_SAFE_CALL( hipMalloc( (void **) &d_A, m*n*sizeof(float) ) );/* Allocate memory for d_A */
  CUDA_SAFE_CALL( hipMalloc( (void **) &d_B, m*n*sizeof(float) ) );/* Allocate memory for d_B */ 

  /* STEP 4: Copy host matrix A to the device matrix d_A */
  CUDA_SAFE_CALL( hipMemcpy( d_A, A, m*n*sizeof(float), hipMemcpyHostToDevice )  ); /* Copy Host matrix A into matrix d_A on Device */

  /* STEP 5: Copy device matrix d_A into device memory d_B */
  CUDA_SAFE_CALL( hipMemcpy( d_B, d_A, m*n*sizeof(float), hipMemcpyDeviceToDevice ) ); /* Copy device matrix d_A into matrix d_B on Device */

  /* STEP 6: Copy back from device memory into the host memory only data corresponding to matrix B (d_B) */
  CUDA_SAFE_CALL( hipMemcpy( B, d_B, m*n*sizeof(float), hipMemcpyDeviceToHost ) ); /* Copy device matrix d_B into host matrix B */

  /* STEP 7: Deallocate device memory */
  CUDA_SAFE_CALL( hipFree(d_A) );/* Deallocate d_A */
  CUDA_SAFE_CALL( hipFree(d_B) ); /* Deallocate d_B */ 

  double error = 0.0;
  for( i=0; i<m; i++ ) {
    for( j=0; j<n; j++ ) {
      error += A(i,j) - B(i,j);
    }
  }
  printf("Error = %f\n",error);

  /* STEP 8: Deallocate host memory */
  free(A);
  free(B);
}

