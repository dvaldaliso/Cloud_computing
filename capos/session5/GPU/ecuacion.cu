
#include <hip/hip_runtime.h>

__global__ void ecuacion(double *solr, double * solim, const double *a, const double *b, const double *c )
{
 int id=threadIdx.x+blockIdx.x*blockDim.x;
double tmp=b[id]*b[id]-4*a[id]*c[id];
 if (tmp>=0)
    {solr[id]=(-b[id]+sqrt(tmp))/(2*a[id]);
     solim[id]=0.0;
    }
else
  {solr[id]=(-b[id]/(2*a[id]));
   solim[id]=(sqrt(-tmp))/(2*a[id]);
  }
}

 
 
