

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;

#define FATAL(msg) \
    do {\
        fprintf(stderr, "[%s:%d] %s\n", __FILE__, __LINE__,(msg));\
        exit(-1);\
    } while(0)

#define min(a,b) ((a)<(b)?(a):(b))
#define BS 256

__global__ void matvec_kernel(int n,double *A,double *x,double *y)
{
  int i,j;
  double res = 0.0;

  // calculate global thread index based on the block and thread indices
  i = blockDim.x*blockIdx.x+threadIdx.x;
  if (i<n) {
    for (j=0; j<n; j++) {
      res += A[i+j*n]*x[j];
    }
    y[i] = res;
  }
}

// Check the result of matvec (in CPU)
void verify(int n,double *A, double *x, double *y)
{
  int i, j;
  double *z,err=0.0,nrm=0.0;

  z = (double*)malloc(n*sizeof(double));
  for (i=0; i<n; i++) {
    z[i] = 0.0;
    for (j=0; j<n; j++) {
      z[i] += A[i+j*n]*x[j];
    }
    err += fabs(z[i]-y[i]);
    nrm += fabs(z[i]);
  }
  if (err/nrm>1e-13) printf(" relative error = %g\n",err/nrm);
  else printf(" PASS\n");
  free(z);
}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

double randvalue()
{
  return (double)((rand()/(double)((unsigned int)RAND_MAX+1)))-0.5;
}

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 1000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./matvec               # Matrix of size 1,000 is used"
           "\n    Usage: ./matvec <m>           # Matrix of size m is used"
           "\n");
        exit(0);
    }

    double* h_A = (double*) malloc( sizeof(double)*n*n );
    for (unsigned int i=0; i < n*n; i++) { h_A[i] = randvalue(); }

    double* h_x = (double*) malloc( sizeof(double)*n );
    for (unsigned int i=0; i < n; i++) { h_x[i] = randvalue(); }

    double* h_y = (double*) malloc( sizeof(double)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Matrix size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    double *d_A = NULL, *d_x = NULL, *d_y = NULL;
    hipMalloc((void**)&d_A, sizeof(double)*n*n);
    hipMalloc((void**)&d_x, sizeof(double)*n);
    hipMalloc((void**)&d_y, sizeof(double)*n);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    hipMemcpy(d_A,h_A,sizeof(double)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(d_x,h_x,sizeof(double)*n,hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    dim3 nthreads(BS);
    //dim3 nblocks((n+nthreads.x-1)/nthreads.x);
    dim3 nblocks(ceil(float(n)/BS));
    matvec_kernel<<<nblocks,nthreads>>>(n,d_A,d_x,d_y);

    cuda_ret = hipGetLastError();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    hipMemcpy(h_y,d_y,sizeof(double)*n,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(n, h_A, h_x, h_y);

    // Free memory ------------------------------------------------------------

    free(h_A);
    free(h_x);
    free(h_y);
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    return 0;

}

