// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// defines
#define N 20
// declaracion de funciones
// GLOBAL: funcion llamada desde el host y ejecutada en el device (kernel)
__global__ void suma_gpu( float *A, float *B, float *C )
{
    // indice de columna
    int columna = threadIdx.x;
    // indice de fila
    int fila = threadIdx.y;
    // indice lineal
    int myID = columna + fila * blockDim.x;
    // sumamos cada elemento
    C[myID] = A[myID] + B[myID];
}
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
    // declaraciones
    float *hst_A, *hst_B, *hst_C;
    float *dev_A, *dev_B, *dev_C;
    // reserva en el host
    hst_A = (float*)malloc(N*N*sizeof(float));
    hst_B = (float*)malloc(N*N*sizeof(float));
    hst_C = (float*)malloc(N*N*sizeof(float));
    
    // reserva en el device
    hipMalloc( (void**)&dev_A, N*N*sizeof(float));
    hipMalloc( (void**)&dev_B, N*N*sizeof(float));
    hipMalloc( (void**)&dev_C, N*N*sizeof(float));
    
    // incializacion
    
    for(int i=0;i<N*N;i++)
    {
        hst_A[i] = (float)( rand() % 10 );
        hst_B[i] = (float)( rand() % 10 );
    }
    // copia de datos
    hipMemcpy( dev_A, hst_A, N*N*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_B, hst_B, N*N*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_C, hst_C, N*N*sizeof(float), hipMemcpyHostToDevice );
    
    // dimensiones del kernel
    dim3 Nbloques(1);
    dim3 hilosB(N,N);
    // llamada al kernel bidimensional de NxN hilos
    suma_gpu<<<Nbloques,hilosB>>>(dev_A, dev_B, dev_C);
    // recogida de datos
    hipMemcpy( hst_C, dev_C, N*N*sizeof(float), hipMemcpyDeviceToHost );

    // impresion de resultados
    printf("A:\n");
     for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            printf("%2.0f ",hst_A[j+i*N]);
        }
        printf("\n");
    }
    printf("B:\n");
     for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            printf("%2.0f ",hst_B[j+i*N]);
        }
        printf("\n");
    }
    printf("C:\n");
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            printf("%2.0f ",hst_C[j+i*N]);
        }
        printf("\n");
    }
    // salida
    printf("\npulsa INTRO para finalizar...");
    fflush(stdin);
    char tecla = getchar();
    return 0;
}