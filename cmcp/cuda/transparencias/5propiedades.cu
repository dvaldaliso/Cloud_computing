// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
hipDeviceProp_t deviceProp;
int deviceID;
hipGetDevice( &deviceID );
hipGetDeviceProperties( &deviceProp,deviceID );
printf("MAX Threads per block: %d\n",deviceProp.maxThreadsPerBlock);
printf("MAX BLOCK SIZE\n");
printf(" [x -> %d]\n [y -> %d]\n [z -> %d]\n",deviceProp.maxThreadsDim[0],
deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
printf("MAX GRID SIZE\n");
printf(" [x -> %d]\n [y -> %d]\n [z -> %d]\n",deviceProp.maxGridSize[0],
deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
// salida
printf("\npulsa INTRO para finalizar...");
fflush(stdin);
char tecla = getchar();
return 0;
}