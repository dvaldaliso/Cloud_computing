// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define N 8
/*
En el siguiente ejemplo se muestran las diferencias y las similitudes que existen a la hora
de reservar memoria tanto en el host como en el device. En este ejemplo se reserva espacio
para una matriz cuadrada de NN elementos, se inicializa en el host con valores aleatorios2
(entre 0 y 9) de tipo float y después se transfieren los datos desde el host hasta el device:
*/
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
// declaracion
 float *hst_matriz;
 float *dev_matriz;
// reserva en el host
 hst_matriz = (float*)malloc( N*N*sizeof(float) );
// reserva en el device
 hipMalloc( (void**)&dev_matriz, N*N*sizeof(float) );
// inicializacion de datos
 srand ( (int)time(NULL) );
 for (int i=0; i<N*N; i++)
 {
 hst_matriz[i] = (float)( rand() % 10 );
 }
// copia de datos
 hipMemcpy(dev_matriz, hst_matriz, N*N*sizeof(float), hipMemcpyHostToDevice);
// salida
 hipFree( dev_matriz );
 printf("\npulsa INTRO para finalizar...");
 fflush(stdin);
 char tecla = getchar();
 return 0;
}