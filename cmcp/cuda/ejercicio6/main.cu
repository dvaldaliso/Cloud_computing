/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    float* h_A = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { h_A[i] = (rand()%100)/100.00; }

    float* h_B = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { h_B[i] = (rand()%100)/100.00; }

    float* h_C = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------
    hipMalloc( (void**)&d_A, n*sizeof(float));
    hipMalloc( (void**)&d_B, n*sizeof(float));
    hipMalloc( (void**)&d_C, n*sizeof(float));

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE












    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE






    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE





    cuda_ret = hipGetLastError();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE



    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(h_A, h_B, h_C, n);

    // Free memory ------------------------------------------------------------

    free(h_A);
    free(h_B);
    free(h_C);

    //INSERT CODE HERE




    return 0;

}

