/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"

// tamano de los vectores
#define BLOCK 5 // tamano del bloque

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    float *d_A, *d_B, *d_C;

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    float* h_A = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { h_A[i] = (rand()%100)/100.00; }

    float* h_B = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { h_B[i] = (rand()%100)/100.00; }

    float* h_C = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    hipMalloc( (void**)&d_A, n*sizeof(float));
    hipMalloc( (void**)&d_B, n*sizeof(float));
    hipMalloc( (void**)&d_C, n*sizeof(float));


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    hipMemcpy(d_A, h_A, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n*sizeof(float), hipMemcpyHostToDevice);


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
     int nBloques = n/BLOCK;
    if (n%BLOCK != 0)
    {
        nBloques = nBloques + 1;
    }
    int hilosB = BLOCK;
    printf("Vector de %d elementos\n", n);
    printf("Lanzamiento con %d bloques (%d hilos)\n", nBloques, nBloques*hilosB);

    vecAddKernel<<< nBloques, hilosB >>>( d_A, d_B, d_C, n);

    cuda_ret = hipGetLastError();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    hipMemcpy(h_C, d_C, n*sizeof(float), hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(h_A, h_B, h_C, n);

    // Free memory ------------------------------------------------------------

    free(h_A);
    free(h_B);
    free(h_C);

    //INSERT CODE HERE
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);



    return 0;

}

