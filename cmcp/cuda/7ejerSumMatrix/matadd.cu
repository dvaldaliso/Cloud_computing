#include "hip/hip_runtime.h"
#include <stdio.h>
#include "support.h"

__global__ void matAddKernel(float* A, float* B, float* C, int n) {

    // Calculate global thread indices based on the block and thread indices ----
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    

    // Use global indices to determine which elements to read, add, and write ---
    if (i<n && j<n) C[i*n+j] = A[i*n+j] + B[i*n+j];

}

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./matadd               # Matrix of size 10,000x10,000 is used"
           "\n    Usage: ./matadd <n>           # Matrix of size n is used"
           "\n");
        exit(0);
    }

    float* h_A = (float*) malloc( sizeof(float)*n*n );
    for (unsigned int i=0; i < n*n; i++) { h_A[i] = (rand()%100)/100.00; }

    float* h_B = (float*) malloc( sizeof(float)*n*n );
    for (unsigned int i=0; i < n*n; i++) { h_B[i] = (rand()%100)/100.00; }

    float* h_C = (float*) malloc( sizeof(float)*n*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Matrix size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    float *d_A = NULL, *d_B = NULL, *d_C = NULL;
    hipMalloc((void**)&d_A, sizeof(float)*n*n);
    hipMalloc((void**)&d_B, sizeof(float)*n*n);
    hipMalloc((void**)&d_C, sizeof(float)*n*n);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    hipMemcpy(d_A, h_A, sizeof(float)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float)*n*n, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    dim3 nhilos(32,32);
    dim3 nbloques(ceil(float(n)/nhilos.x),ceil(float(n)/nhilos.y));
    matAddKernel<<<nbloques,nhilos>>>(d_A, d_B, d_C, n);

    cuda_ret = hipGetLastError();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    hipMemcpy(h_C, d_C, sizeof(float)*n*n, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(h_A, h_B, h_C, n);

    // Free memory ------------------------------------------------------------

    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}

